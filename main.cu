
#include "hip/hip_runtime.h"
#include ""

#include <vector>
#include <iostream>

#include <stdio.h>


#include "NeuralNetwork.h"
#include "Activation.h"
#include "Error.h"


#include <iomanip>
#include <windows.h>

/*The MIT License (MIT)
Copyright � 2022 Alexander Joseph Kostyak

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the �Software�),
to deal in the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute,
sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED �AS IS�, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.*/

//Development credit to Ryan Wise GitHub user rdw88 for substantial codebase:
//https://github.com/rdw88/CUDA-Neural-Network

int main(int argc, char** argv)
{
	for (int i = 0; i < argc; ++i)
		std::cout << argv[i] << "\n";

	const unsigned int inputlayersize = 10;
	const unsigned int outputlayersize = 12;
	const unsigned int batchsizearg = 1;
	const float learningrate = 0.1;


	std::vector<int> layers;


	
	//activation parameters, as types and bests with exe argument and output validation;
	//function to csv of sorts

	NeuralNetwork network = NeuralNetwork({ inputlayersize, 50, 50, outputlayersize }, batchsizearg, learningrate);

	Activation act1 = newActivation(RELU);
	Activation act2 = newActivation(RELU);
	Activation act3 = newActivation(RELU);
	act1.maxThreshold = 5;
	act2.maxThreshold = 10;
	act3.maxThreshold = 15;
	act1.leakyReluGradient = 0.01;
	act2.leakyReluGradient = 0.01;
	act3.leakyReluGradient = 0.01;


	//generate executable that accepts necessary parameters (net name or filename / constructor filename, operations, etc...)

	network.setLayerActivations({ act1,		act2,		act3,		newActivation(SIGMOID) });

	network.setLossFunction(MEAN_SQUARED_ERROR);

	//all input/output as float from 0-1

	std::vector<float> input;
	//std::vector<float> output;




	std::vector<float> single_input { 0.15, 0.45, .78, 0.04, 0.45, 0.73, 1, 0.11, 0.01, 0.11 };
	std::vector<float> single_input_2 {0.38, 0.92, 0.16, 0.63, 0.82, 0.11, .2, 0.73, 0.25, 0.68};
	
	std::vector<float> input_values{ 0.15, 0.45, .78, 0.04, 0.45, 0.73, 1, 0.11, 0.01, 0.11 };
	std::vector<float> input_values_2{ 0.38, 0.92, 0.16, 0.63, 0.82, 0.11,.2, 0.73, 0.25, 0.68 };

	std::vector<float> output_values { 0.1, 0.2, 0.3, 0.4, 0.5, 0.6, 0.7, 0.8, .9, 1.0, 0.9, 1.0 };            
	std::vector<float> output_values_2 { 0.3, 0.9, 0.5, 0.0, 0.7, 0.4, 0.4, 0.1, 0.2, 0.3, 0.8, 0.3 };           


	std::vector<float> output_valuest{ 0.1, 0.2, 0.3, 0.4, 0.5, 0.6, 0.7, 0.8,.9, 1.0 , 0.9, 1.0 };            
	std::vector<float> output_values_2t{ 0.3, 0.9, 0.5, 0.0, 0.7, 0.4, 0.4, 0.1,0.2, 0.3, 0.8, 0.3 };           


	
	//create input batch
	for (int i =0; i< batchsizearg * inputlayersize - inputlayersize; i++)
	{
		input_values.push_back(single_input[i % inputlayersize]);
		input_values_2.push_back(single_input_2[i % inputlayersize]);
	}

	//create output batch
	for (int i = 0; i < batchsizearg * outputlayersize - outputlayersize; i++)
	{
		output_values.push_back(output_valuest[i % outputlayersize]);
		output_values_2.push_back(output_values_2t[i % outputlayersize]);
	}

	std::cout << std::fixed;
	std::cout << std::setprecision(6);

	long int before = GetTickCount();


	std::cout << std::endl << std::endl << "batch size:   " << network.getBatchSize() << std::endl << "input layer size:   " << "should be 10" << std::endl << std::endl;
	for (int i = 0; i < 500; i++)
	{
		network.train(input_values, output_values);
		network.train(input_values_2, output_values_2);
		std::cout << " " << network.getTotalError()[0];
	}

	long int after = GetTickCount();

	std::cout << std::endl << "time elapsed ms:   " << after - before << std::endl;


	std::cout << std::endl;
			

	std::vector<float> output = network.getOutputForInput(single_input);
	//get_output(single_input)

	for (int i = 0; i < outputlayersize; i++) // (int i in output)
	{
		std::cout  << single_input[i%10] << "   " << output_values[i] << "   " << output[i] << std::endl ;
	}
			//for i, item in enumerate(output) :
				//print('%.1f' % item, output_values[i])

	std::cout << "---------------------------------" << std::endl;



	output = network.getOutputForInput(single_input_2);
	for (int i = 0; i < outputlayersize; i++) // (int i in output)
	{
		std::cout << single_input_2[i%10] << "   " << output_values_2[i] << "   " << output[i] << std::endl;
	}

	std::cin.get();



    return 0;
}